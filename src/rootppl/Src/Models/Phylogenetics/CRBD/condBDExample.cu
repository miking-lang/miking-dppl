#include "hip/hip_runtime.h"
#include "../../../Inference/Smc/smcImpl.cuh"

#include "../TreeUtils/treeUtils.cuh"
#include "simulations.cuh"
#include "stack.cuh"

/**
    This model traverses the tree with a DFS path that corresponds to the recursive calls. 
*/

// nvcc -arch=sm_75 -rdc=true Src/Models/Phylogenetics/CRBD/condBDExample.cu -o smc.exe -lcudadevrt -std=c++11 -O3 -D GPU

// Compile CPU: g++ -x c++ Src/Models/Phylogenetics/CRBD/condBDExample.cu -o smc.exe -std=c++11 -O3

#define NUM_BBLOCKS 5
INIT_GLOBAL(pStack_t, NUM_BBLOCKS)


BBLOCK_DATA(tree, tree_t, 1)

struct progState_t {
    floating_t lambda;
    floating_t mu;
    int treeIdx;
    int parentIdx;
};

BBLOCK_HELPER(pushChild, {

    progState_t pState;
    pState.parentIdx = parent.treeIdx;
    pState.treeIdx = childIdx;
    pState.lambda = parent.lambda;
    pState.mu = parent.mu;
    PSTATE.pushType(pState);

}, void, progState_t parent, int childIdx)


BBLOCK(condBD_1, pStack_t, {

    tree_t* treeP = DATA_POINTER(tree);

    progState_t pState;
    PSTATE.popType<progState_t>(&pState);
    PSTATE.pushType<progState_t>(pState);

    floating_t parentAge = treeP->ages[pState.parentIdx];
    floating_t treeAge = treeP->ages[pState.treeIdx];
    WEIGHT(- pState.mu * (parentAge - treeAge));

    PC++; // condBD_2
    // Resamples here
})

BBLOCK(condBD_2, pStack_t, {
    
    tree_t* treeP = DATA_POINTER(tree);

    progState_t pState;
    PSTATE.popType<progState_t>(&pState);

    floating_t parentAge = treeP->ages[pState.parentIdx];
    floating_t treeAge = treeP->ages[pState.treeIdx];
    floating_t w = BBLOCK_CALL(simBranch<pStack_t>, parentAge, treeAge, pState.lambda, pState.mu);
    WEIGHT(w);
    

    if(treeP->idxLeft[pState.treeIdx] != -1) { // Interior node, keep DFSing
        WEIGHT(log(2.0 * pState.lambda));
        PC++; // condBD_3
        PSTATE.pushType<progState_t>(pState);
        // Resamples here
    } else {
        PC = PSTATE.pop();
        if (PC < NUM_BBLOCKS) { // Does not resample here
            // pplFunc_t<pStack_t> funcToCall = DATA_POINTER(bblocksArr)[PC];
            // BBLOCK_CALL(funcToCall, NULL);
            BBLOCK_CALL(DATA_POINTER(bblocksArr)[PC], NULL);
        }
    }
})

// Keep DFSing left
BBLOCK(condBD_3, pStack_t, {

    tree_t* treeP = DATA_POINTER(tree);

    progState_t pState;
    PSTATE.popType<progState_t>(&pState);

    PSTATE.pushType<progState_t>(pState); // Need to use this state again when exploring right sibling
    PSTATE.push(4); // PC = condBD_4

    int leftIdx = treeP->idxLeft[pState.treeIdx];
    BBLOCK_CALL(pushChild, pState, leftIdx);

    PC = 1;
    BBLOCK_CALL(condBD_1);
})

// Keep DFSing right
BBLOCK(condBD_4, pStack_t, {

    tree_t* treeP = DATA_POINTER(tree);

    progState_t pState;
    PSTATE.popType<progState_t>(&pState);

    // What should the return address be here? Do I really need one?

    int rightIdx = treeP->idxRight[pState.treeIdx];
    BBLOCK_CALL(pushChild, pState, rightIdx);

    PC = 1;
    BBLOCK_CALL(condBD_1);
})

BBLOCK(condBD_init, pStack_t, {

    //PSTATE.arr = DATA_POINTER(globalStack);
    //PSTATE.stackPointer = i;
    //printf("arr: %p, sp: %d\n", PSTATE.arr, PSTATE.stackPointer);

    //printf("sp: %d\n", PSTATE.stackPointer);
    PSTATE.push(NUM_BBLOCKS); // Go to PC=5 when top-level condBD_1 is done (terminate)

    progState_t pState;
    pState.parentIdx = -1;
    pState.treeIdx = ROOT_IDX;
    pState.lambda = 0.2;
    pState.mu = 0.1;
    PSTATE.pushType<progState_t>(pState);
    
    WEIGHT(log(2.0));

    PC = 3;
    BBLOCK_CALL(condBD_3);
})

MAIN(

    INITBBLOCK(condBD_init, pStack_t)
    INITBBLOCK(condBD_1, pStack_t)
    INITBBLOCK(condBD_2, pStack_t)
    INITBBLOCK(condBD_3, pStack_t)
    INITBBLOCK(condBD_4, pStack_t)

    SMC(pStack_t, NULL)
)
