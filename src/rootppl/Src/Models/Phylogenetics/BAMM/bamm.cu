#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include "../../Inference/Smc/smc.cuh"
#include "../../Inference/Smc/smcImpl.cuh"
#include "../../Utils/distributions.cuh"
#include "TreeUtils/treeUtils.cuh"
#include "bamm.cuh"

// nvcc -arch=sm_75 -rdc=true Src/Models/Phylogenetics/BAMM/bamm.cu Src/Utils/*.cpp -o smc.exe -lcudadevrt -std=c++11 -O3 -D GPU

// Compile CPU: g++ -x c++ Src/Models/Phylogenetics/BAMM/bamm.cu Src/Utils/*.cpp -o smc.exe -std=c++11 -O3

#define MIN(a, b) a <= b ? a : b
#define MAX(a, b) a >= b ? a : b

BBLOCK_DATA(tree, tree_t, 1);
// BBLOCK_DATA(lambda, floating_t, 1) // prolly faster to just pass these as args... they should be generated in particle anyway?
// BBLOCK_DATA(mu, floating_t, 1)

// BBLOCK_DATA(nestedArgs, bblockArgs_t, NUM_PARTICLES)


floating_t corrFactor;


void initBamm() {
    // lambda ~ gamma( 1.0, 1.0 )
    // mu     ~ gamma( 1.0, 1.0 )
    // *lambda = 0.2; // birth rate
    // *mu = 0.1; // death rate

    int numLeaves = countLeaves(tree->idxLeft, tree->idxRight, NUM_NODES);
    corrFactor = (numLeaves - 1) * log(2.0) - lnFactorial(numLeaves);

    COPY_DATA_GPU(tree, tree_t, 1);
    // COPY_DATA_GPU(lambda, floating_t, 1)
    // COPY_DATA_GPU(mu, floating_t, 1)

    //COPY_DATA_GPU(nestedArgs, bblockArgs_t, NUM_PARTICLES)
}

DEV HOST inline floating_t lambdaFun(lambdaFun_t lf, floating_t t) {
    return lf.lambda * exp(lf.z * (lf.t1 - t));
}


BBLOCK_HELPER(lambdaWait, {

    //printf("Will write lambda vals...\n");
    floating_t startLambda = lambdaFun(lf, startTime);
    floating_t stopLambda = lambdaFun(lf, stopTime);

    floating_t topLambda = MAX(startLambda, stopLambda);
    //printf("Wrote lambda vals!\n");

    floating_t t = startTime - BBLOCK_CALL(sampleExponential, topLambda);
    // printf("Made exponential call!\n");

    if(t < stopTime || BBLOCK_CALL(flipK, lambdaFun(lf, t) / topLambda))
        return startTime - t;

    return startTime - BBLOCK_CALL(lambdaWait, lf, t, stopTime);

}, floating_t, lambdaFun_t lf, floating_t startTime, floating_t stopTime)

// Forward simulation from a starting time, returning extinction (true) or survival (false)
BBLOCK_HELPER(goesExtinct, {

    //if(recursionCount > 80)
        //printf("RecursionCount: %d\n", recursionCount);

    floating_t t1 = BBLOCK_CALL(sampleExponential, mu + sigma);
    floating_t tLambda = BBLOCK_CALL(lambdaWait, lf, startTime, 0);

    floating_t t = MIN(t1, tLambda);

    floating_t currentTime = startTime - t;
    if(currentTime < 0)
        return false;
    
    if(t1 < tLambda) {
        bool extinction = BBLOCK_CALL(flipK, mu / (mu+sigma));
        if (extinction)
            return true;

        // No extinction, so rateshift
        
        floating_t lambda2 = BBLOCK_CALL(sampleGamma, 1.0, 1.0);
        floating_t z2 = BBLOCK_CALL(sampleNormal, 0.0, 0.001);
        floating_t mu2 = BBLOCK_CALL(sampleGamma, 1.0, 1.0);
        lambdaFun_t lf2(lambda2, z2, t1);
        
        return BBLOCK_CALL(goesExtinct<T>, currentTime, lf2, mu2, sigma, recursionCount+1);
        // return BBLOCK_CALL(goesExtinct<T>, currentTime, lambdaFun_t{ BBLOCK_CALL(gamma, 1.0, 1.0) , BBLOCK_CALL(normal, 0.0, 0.001) , t1}, BBLOCK_CALL(gamma, 1.0, 1.0) , sigma);
    }

    return BBLOCK_CALL(goesExtinct<T>, currentTime, lf, mu, sigma, recursionCount+1)
        && BBLOCK_CALL(goesExtinct<T>, currentTime, lf, mu, sigma, recursionCount+1);

}, bool, floating_t startTime, lambdaFun_t lf, floating_t mu, floating_t sigma, int recursionCount = 0)


BBLOCK(goesExtinctBblock, nestedProgState_t, {
    tree_t* treeP = DATA_POINTER(tree);
    double age = treeP->ages[ROOT_IDX];
    bblockArgs_t params = *static_cast<bblockArgs_t*>(arg);
    
    PSTATE.extinct = BBLOCK_CALL(goesExtinct<nestedProgState_t>, age, params.lf, params.mu, params.sigma);
    PC++;
    RESAMPLE = true;
})


BBLOCK_HELPER(simBranch, {

    floating_t tLambda = BBLOCK_CALL(lambdaWait, lf, startTime, stopTime);

    // floating_t tSigma = BBLOCK_CALL(sampleExponential, sigma)
    floating_t tSigma = tLambda + 1.0; // For testing
    floating_t t = MIN(tLambda, tSigma);

    floating_t currentTime = startTime - t;

    // This is the terminating condition, return final values and appropriate accumulators
    // for number of shifts and probability
    if (currentTime <= stopTime) {
        floating_t t1 = startTime - stopTime;
        floating_t meanLambda = (lambdaFun(lf, startTime) + lambdaFun(lf, stopTime)) / 2.0;
        
        simBranchRet_t rt(lf, z, mu, meanLambda*t1, z*t1, mu*t1, 0, -mu*t1);
        return rt;
    }

    // Check whether this is a rate shift, and handle this case
    if (tSigma < tLambda) {
        floating_t lambda0_2 = BBLOCK_CALL(sampleGamma, 1.0, 1.0);
        floating_t z2 = BBLOCK_CALL(sampleNormal, 0.0, 0.001);
        floating_t mu2 = BBLOCK_CALL(sampleGamma, 1.0, 1.0);

        // Recursive call
        lambdaFun_t lf2(lambda0_2, z2, currentTime);
        simBranchRet_t ret = BBLOCK_CALL(simBranch<T>, currentTime, stopTime, lf2, z2, mu2, sigma);

        // Return accumulated values
        // Add this rate shift to the total number of rate shifts
        // No-extinction probability is dependent on mu before shift
        floating_t meanLambda = (lambdaFun(lf, startTime) + lambdaFun(lf, currentTime)) / 2.0;
        
        simBranchRet_t rt(ret.lf, ret.r1, ret.r2, ret.r3 + meanLambda*t, ret.r4 + z*t, ret.r5 + mu*t, ret.r6 + 1, ret.r7 - mu*t);
        return rt;
    }

    // We have a speciation event; handle this case
    bool sideExtinction = BBLOCK_CALL(goesExtinct<T>, currentTime, lf, mu, sigma);
    if (sideExtinction == false) {
        simBranchRet_t rt(lf, 0.0, 0.0, 0.0, 0.0, 0.0, 0, -INFINITY);
        return rt;
    }

    // Recursive call
    simBranchRet_t ret = BBLOCK_CALL(simBranch<T>, currentTime, stopTime, lf, z, mu, sigma);

    // Return accumulated values
    // Factor 2 because we do not care whether extinction is on left or right side branch
    floating_t meanLambda = (lambdaFun(lf, startTime) + lambdaFun(lf, currentTime)) / 2.0;

    simBranchRet_t rt(ret.lf, ret.r1, ret.r2, ret.r3 + meanLambda*t, ret.r4 + z*t, ret.r5 + mu*t, ret.r6, ret.r7 + log(2.0) - mu*t);
    return rt;

}, simBranchRet_t, floating_t startTime, floating_t stopTime, lambdaFun_t lf, floating_t z, floating_t mu, floating_t sigma)


// TODO: Should return tree info as string?
BBLOCK(simTree, progState_t, {

    // Fetch tree data
    tree_t* treeP = DATA_POINTER(tree);
    treeIdx_t treeIdx = PSTATE.treeIdx;
    PSTATE.treeIdx = treeP->idxNext[treeIdx];

    // Terminate if tree is fully traversed
    if(treeIdx == -1) {
        PC++;
        RESAMPLE = false;
        return;
    }

    int indexParent = treeP->idxParent[treeIdx];

    if(indexParent == -1) {
        // Recursive call under diversification rates
        
        // Return total subtree with diversification info attached as an 'extended newick' character string

        // Root seems to only recurse and conclude tree info?
        bblockArgs_t args = PSTATE.stack.peek();
        PSTATE.stack.push(args);
        RESAMPLE = false;
        return;
    }

    bblockArgs_t args = PSTATE.stack.pop();


    floating_t treeAge = treeP->ages[treeIdx];
    simBranchRet_t ret = BBLOCK_CALL(simBranch<progState_t>, treeP->ages[indexParent], treeAge, args.lf, args.lf.z, args.mu, args.sigma);

    bool interiorNode = treeP->idxLeft[treeIdx] != -1 || treeP->idxRight[treeIdx] != -1;
    floating_t lnSpecProb = interiorNode ? log(lambdaFun(args.lf, treeAge)) : 0.0;
    WEIGHT(ret.r7 + lnSpecProb);

    // Collect node info
    
    // Collect branch info

    RESAMPLE = true;
    if(interiorNode) {
        bblockArgs_t args2(ret.lf, ret.r2, args.sigma);
        PSTATE.stack.push(args2);
        PSTATE.stack.push(args2);

        // Recursive call under final diversification rates (done automatically by particle PC framework)

        // Return total subtree with diversification info attached as an 'extended newick' character string
        return;
    } else {
        // Return leaf with diversification info attached as an 'extended newick' character string
        return;
    }

})


BBLOCK(simBAMM, progState_t, {
    tree_t* treeP = DATA_POINTER(tree);

    PSTATE.treeIdx = treeP->idxLeft[ROOT_IDX];

    //    var lambda_0    = DistLambda.sample()
    //    var z_0         = DistZ.sample()
    //    var lambdaFun_0 = function( t ) { lambda_0 * Math.exp( z_0 * (tree.age - t ) ) }
    //    var mu_0        = DistMu.sample()
    //    var sigma       = gamma( {shape:1, scale:1} )

    // Test settings
    floating_t lambda_0 = 0.2;
    floating_t z_0 = 0.0;
    floating_t age = treeP->ages[ROOT_IDX];
    floating_t mu_0 = 0.1;
    floating_t sigma = 0.000001;

    lambdaFun_t lf(lambda_0, z_0, age);
    bblockArgs_t args(lf, mu_0, sigma);

    // one for each child, and one for nested inference after tree simulations
    /*
    for(int i = 0; i < 3; i++)
        PSTATE.stack.push(args);
    */
    PSTATE.stack.push(args);
    PSTATE.stack.push(args);
    PSTATE.stack.push(args);

    PC++;
    RESAMPLE = false;
})

CALLBACK(calcResult, nestedProgState_t, {
    int numExtinct = 0;
    for(int i = 0; i < NUM_PARTICLES_NESTED; i++)
        numExtinct += PSTATE.extinct;

    int numSurvived = NUM_PARTICLES_NESTED - numExtinct;
    return_t* retP = static_cast<return_t*>(ret);
    *retP = numSurvived / (double)NUM_PARTICLES_NESTED;
    
}, void* ret)

template <typename T>
DEV T runNestedInference(int parentIndex, bblockArgs_t* arg) {
    bool parallelExec = false, parallelResampling = false;

    T ret;

    SMCSTART(nestedProgState_t)

    INITBBLOCK_NESTED(goesExtinctBblock, nestedProgState_t)
    
    SMCEND_NESTED(nestedProgState_t, calcResult, ret, arg, parallelExec, parallelResampling, parentIndex)

    return ret;
}

BBLOCK(survivalConditioning, progState_t, {
    bblockArgs_t args = PSTATE.stack.pop();
    bblockArgs_t* argsAlloc = new bblockArgs_t;
    *argsAlloc = args;
    double survivalRate = runNestedInference<double>(i, argsAlloc);
    free(argsAlloc);

    WEIGHT(-2.0 * log(survivalRate));

    PC++;
    RESAMPLE = false;
})

STATUSFUNC({
    
})

int main() {

    initGen();
    initBamm();
    
    SMCSTART(progState_t)

    INITBBLOCK(simBAMM, progState_t)
    INITBBLOCK(simTree, progState_t)
    INITBBLOCK(survivalConditioning, progState_t)

    SMCEND(progState_t)

    res += corrFactor;  

    cout << "log(MarginalLikelihood) = " << res << endl;

    return 0;
}

