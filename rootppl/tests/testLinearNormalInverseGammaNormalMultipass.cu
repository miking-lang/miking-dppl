#include "hip/hip_runtime.h"
/**
 * Tests waitingTimeDelayed
 */

#include <iostream>
#include <cstring>
#include <string>
#include <iostream>
#include <fstream>

#include "inference/smc/smc.cuh"
#include "utils/math.cuh"
#include "dists/delayed.cuh"

const floating_t m0 = 0.1; // mean
const floating_t v = 0.9; // variance multiplier
const floating_t a = 10; // scale
const floating_t b = 0.2; // scale
const int passes = 100;

const std::string testName = "testLinearNormalInverseGammaNormalMultipass";

int numParts; // number of particles, supplied by first argument
int numRuns; // number of runs supplied by the command line

INIT_MODEL(floating_t);

BBLOCK(testNormalInverseGammaNormal, {
  /* We will sample two waiting times (so that we have an update on the rate),
     and then check the distribution of the second waiting time against WebPPL.*/
    normalInverseGamma_t prior(m0, v, a, b);
    
    floating_t statistic = SAMPLE(linearNormalInverseGammaNormal, prior, 1.0, 0.0, 1.0);
    floating_t statistic2 = SAMPLE(linearNormalInverseGammaNormal, prior, 1.0, 0.0, 1.0);
    floating_t statistic3 = SAMPLE(linearNormalInverseGammaNormal, prior, 1.0, 0.0, 1.0);
    for (int i = 0; i < passes; i++) {
      statistic3 = SAMPLE(normalInverseGammaNormal, prior);
    }
    
    // TODO do it with SAMPLE, instead of invoking the function directly
       
    PSTATE = statistic3;
    NEXT=NULL;
});

CALLBACK(stats, {
    std::string fileName = "tests/" + testName + ".csv";
    std::ofstream resultFile (fileName, std::ios_base::app);
    if(resultFile.is_open()) {
      for(int i = 0; i < N; i++) {
	resultFile << PSTATES[i] << ", " << exp(WEIGHTS[i])/numRuns << "\n";
      }
      resultFile.close();
    } else {
      printf("Couldnot open file %s\n", fileName.c_str());
    }
})

MAIN({
    if(argc > 2) { 
      numRuns = atoi(argv[2]);			
    }
    else {
      numRuns = 1;
    }
    
    FIRST_BBLOCK(testNormalInverseGammaNormal);
  
    SMC(stats);
  })
