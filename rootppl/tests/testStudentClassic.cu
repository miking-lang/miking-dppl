#include "hip/hip_runtime.h"
/**
 * Tests waitingTimeDelayed
 */

#include <iostream>
#include <cstring>
#include <string>
#include <iostream>
#include <fstream>

#include "inference/smc/smc.cuh"
#include "utils/math.cuh"
#include "dists/delayed.cuh"

const floating_t k = 13; // degrees of freedom
const std::string testName = "testStudentClassic";

int numParts; // number of particles, supplied by first argument
int numRuns; // number of runs supplied by the command line


INIT_MODEL(floating_t);

BBLOCK(testStudentClassic, {
  /* We will sample two waiting times (so that we have an update on the rate),
     and then check the distribution of the second waiting time against WebPPL.*/
  floating_t statistic = SAMPLE(student_t, k);
       
  PSTATE = statistic;
  NEXT = NULL;
});




CALLBACK(stats, {
    std::string fileName = "tests/" + testName + ".csv";
    std::ofstream resultFile (fileName, std::ios_base::app);
    if(resultFile.is_open()) {
      for(int i = 0; i < N; i++) {
	resultFile << PSTATES[i] << ", " << exp(WEIGHTS[i])/numRuns << "\n";
      }
      resultFile.close();
    } else {
      printf("Couldnot open file %s\n", fileName.c_str());
    }
})



MAIN({
    if(argc > 2) { 
      numRuns = atoi(argv[2]);			
    }
    else {
      numRuns = 1;
    }
    
    FIRST_BBLOCK(testStudentClassic);
  
    SMC(stats);
  })
