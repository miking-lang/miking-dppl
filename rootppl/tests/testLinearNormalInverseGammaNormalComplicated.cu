#include "hip/hip_runtime.h"
/**
 * Tests waitingTimeDelayed
 */

#include <iostream>
#include <cstring>
#include <string>
#include <iostream>
#include <fstream>

#include "inference/smc/smc.cuh"
#include "utils/math.cuh"
#include "dists/delayed.cuh"

const floating_t m0 = 0.1; // mean
const floating_t v = 0.9; // variance multiplier
const floating_t a = 10; // scale
const floating_t b = 0.2; // scale
const floating_t aleph = 0.32; // mu-scale
const floating_t c = 1.5; // mu-shift
const floating_t s2 = 0.32; // variance scale

const std::string testName = "testLinearNormalInverseGammaNormalComplicated";

int numParts; // number of particles, supplied by first argument
int numRuns; // number of runs supplied by the command line

INIT_MODEL(floating_t);

BBLOCK(testNormalInverseGammaNormal, {
  /* We will sample two waiting times (so that we have an update on the rate),
     and then check the distribution of the second waiting time against WebPPL.*/
    normalInverseGamma_t prior(m0, v, a, b);
    
    floating_t statistic = SAMPLE(linearNormalInverseGammaNormal, prior, aleph, c, s2 );
  
    // TODO do it with SAMPLE, instead of invoking the function directly
       
    PSTATE = statistic;
    NEXT = NULL;
});

CALLBACK(stats, {
    std::string fileName = "tests/" + testName + ".csv";
    std::ofstream resultFile (fileName, std::ios_base::app);
    if(resultFile.is_open()) {
      for(int i = 0; i < N; i++) {
	resultFile << PSTATES[i] << ", " << exp(WEIGHTS[i])/numRuns << "\n";
      }
      resultFile.close();
    } else {
      printf("Couldnot open file %s\n", fileName.c_str());
    }
})

MAIN({
    if(argc > 2) { 
      numRuns = atoi(argv[2]);			
    }
    else {
      numRuns = 1;
    }
    
    FIRST_BBLOCK(testNormalInverseGammaNormal);
  
    SMC(stats);
  })
