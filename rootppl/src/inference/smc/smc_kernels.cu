
#include <hip/hip_runtime.h>

/*
 * File smc_kernels.cu contains definitions of kernels used by SMC.
 */

 #ifdef __NVCC__

 #include <curand_kernel.h>
 #include "inference/smc/smc.cuh"
 #include "smc_kernels.cuh"

__global__ void initCurandStates(curandState* randStates, int numThreads, int seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= numThreads || i < 0) return;

    // Double check this seed, need only to be unique over one inference, as time should vary between inferences.
    // curand_init(1234 + clock64(), seed * numThreads + i, 0, &particles->randStates[i]);
    curandState randStateLocal = randStates[i];
    #ifdef SEED
    curand_init(1234 + SEED, seed * numThreads + i, 0, &randStateLocal);
    #else
    curand_init(1234 + clock64(), seed * numThreads + i, 0, &randStateLocal);
    #endif
    randStates[i] = randStateLocal;
}

__global__ void initParticlesNext(particles_t particles, int numParticles, pplFunc_t firstBblock) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= numParticles || i < 0) return;

    particles.next[i] = firstBblock;

}

__global__ void execFuncs(curandState* randStates, particles_t particles, int numParticles, int numThreads, void* arg) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if(i >= numParticles || i < 0) return;
    if(i >= numThreads || i < 0) return;

    curandState randStateLocal = randStates[i];

    for(int j = i; j < numParticles; j += numThreads) {
        // funcs[particles.pcs[i]](&randStateLocal, particles, i, arg);
        // int pc = particles.pcs[j];
        pplFunc_t next = particles.next[j];
        if(next != NULL)
            next(&randStateLocal, particles, j, arg);
        // if(pc < numBblocks && pc >= 0)
            // funcs[pc](&randStateLocal, particles, j, arg);
    }

    randStates[i] = randStateLocal;
}

#endif
