#include "hip/hip_runtime.h"
/*
 * File crbd.cu defines an older variant of the constant 
 * rate birth death model with nested inference. 
 *
 * This model traverses the tree with a pre-computed DFS path (defined by the next 
 * pointer in the tree) that corresponds to the recursive calls in the originial model. 
 */


#include <iostream>
#include <cstring>
#include "inference/smc/smc.cuh"
#include "../tree-utils/tree_utils.cuh"
#include "utils/math.cuh"


typedef short treeIdx_t;
struct progState_t {
    treeIdx_t treeIdx;
};

struct nestedProgState_t {
    bool extinct;
};
typedef double return_t;


#define NUM_BBLOCKS 2
INIT_MODEL(progState_t, NUM_BBLOCKS)
#define NUM_BBLOCKS_NESTED 1

typedef primate_tree_t tree_t;
BBLOCK_DATA(tree, tree_t, 1)

BBLOCK_DATA(lambda, floating_t, 1) // prolly faster to just pass these as args... they should be generated in particle anyway?
BBLOCK_DATA(mu, floating_t, 1)

floating_t corrFactor;


void initCBD() {
    // lambda ~ gamma( 1.0, 1.0 )
    // mu     ~ gamma( 1.0, 1.0 )
    *lambda = 0.2; // birth rate
    *mu = 0.1; // death rate

    int numLeaves = countLeaves(tree->idxLeft, tree->idxRight, tree->NUM_NODES);
    corrFactor = (numLeaves - 1) * log(2.0) - lnFactorial(numLeaves);

    COPY_DATA_GPU(lambda, floating_t, 1)
    COPY_DATA_GPU(mu, floating_t, 1)

}

DEV bool goesExtinct(RAND_STATE_DECLARE floating_t startTime) {
// BBLOCK_HELPER(goesExtinct, {

    floating_t lambdaLocal = *DATA_POINTER(lambda);
    floating_t muLocal = *DATA_POINTER(mu);

    floating_t t = SAMPLE(exponential, lambdaLocal + muLocal);
    
    floating_t currentTime = startTime - t;
    if(currentTime < 0)
        return false;
    
    bool speciation = SAMPLE(bernoulli, lambdaLocal / (lambdaLocal + muLocal));
    if (! speciation)
        return true;
    else 
        return goesExtinct(RAND_STATE currentTime) && goesExtinct(RAND_STATE currentTime);

// }, bool, floating_t startTime)
}



BBLOCK(goesExtinctBblock, nestedProgState_t, {
    tree_t* treeP = DATA_POINTER(tree);
    double age = treeP->ages[ROOT_IDX];
    
    PSTATE_TYPE(nestedProgState_t).extinct = goesExtinct(RAND_STATE age);
    PC++;
})


BBLOCK_HELPER(simBranch, {

    floating_t lambdaLocal = *DATA_POINTER(lambda);

    floating_t t = SAMPLE(exponential, lambdaLocal);

    floating_t currentTime = startTime - t;

    if(currentTime <= stopTime)
        return 0.0;
    
    bool sideExtinction = goesExtinct(RAND_STATE currentTime);
    if(! sideExtinction)
        return -INFINITY;

    // WEIGHT(log(2.0)); // was previously done above survival call, no reason to do it before though (unless resample occurrs there)
    
    return BBLOCK_CALL(simBranch, currentTime, stopTime) + log(2.0);

}, floating_t, floating_t startTime, floating_t stopTime)



BBLOCK(simTree, progState_t, {

    tree_t* treeP = DATA_POINTER(tree);
    int treeIdx = PSTATE.treeIdx;

    if(treeIdx == -1) {
        PC = 2;
        // RESAMPLE = false;
        return;
    }

    int indexParent = treeP->idxParent[treeIdx];

    // Weight here only on once, if root has right child
    /*
    if(treeIdx == 2 && indexParent == ROOT_IDX)
        WEIGHT(log(2.0));
    */
    //WEIGHT(log(*(DATA_POINTER(lambda)))); 

    floating_t parentAge = treeP->ages[indexParent];
    floating_t age = treeP->ages[treeIdx];
    

    floating_t lnProb1 = - (*DATA_POINTER(mu)) * (parentAge - age);

    // Interior if at least one child
    bool interiorNode = treeP->idxLeft[treeIdx] != -1 || treeP->idxRight[treeIdx] != -1;
    floating_t lnProb2 = interiorNode ? log(*DATA_POINTER(lambda)) : 0.0;

    floating_t lnProb3 = BBLOCK_CALL(simBranch, parentAge, age);

    /*
    if(treeP->idxLeft[treeIdx] != -1) { // If left branch exists, so does right..
        WEIGHT(log(*DATA_POINTER(lambda)));
    */

    WEIGHT(lnProb1 + lnProb2 + lnProb3);
    PSTATE.treeIdx = treeP->idxNext[treeIdx];
    // RESAMPLE = true;

})

CALLBACK_NESTED(calcResult, nestedProgState_t, {
    int numExtinct = 0;
    for(int i = 0; i < N; i++)
        numExtinct += PSTATES_TYPE(nestedProgState_t)[i].extinct;

    int numSurvived = N - numExtinct;
    return_t* retP = static_cast<return_t*>(ret);
    *retP = numSurvived / (double)N;
    
}, void* ret)

template <typename T>
DEV T runNestedInference(RAND_STATE_DECLARE int parentIndex) {
    bool parallelExec = false, parallelResampling = false;

    T ret;

    SMC_PREPARE_NESTED(nestedProgState_t, NUM_BBLOCKS_NESTED)

    ADD_BBLOCK_NESTED(goesExtinctBblock, nestedProgState_t)
    
    SMC_NESTED(nestedProgState_t, 100, parallelExec, parallelResampling, parentIndex, calcResult, ret, NULL)

    return ret;
}

// Correction Factor moved out to main
BBLOCK(simCRBD, progState_t, {

    tree_t* treeP = DATA_POINTER(tree);

    PSTATE.treeIdx = treeP->idxLeft[ROOT_IDX];

    double survivalRate = runNestedInference<double>(RAND_STATE particleIdx);

    WEIGHT(-2.0 * log(survivalRate));

    PC++;
    // PC = 2;
    BBLOCK_CALL(simTree);
})


MAIN(
    initCBD();
    
    ADD_BBLOCK(simCRBD, progState_t)
    ADD_BBLOCK(simTree, progState_t)

    SMC(NULL)

    res += corrFactor;
)

