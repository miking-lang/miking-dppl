#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include "inference/smc/smc_impl.cuh"
#include "../tree-utils/tree_utils.cuh"

/**
    This file traverses the tree with a precomputed DFS path that corresponds to the recursive calls. 
*/

/*
Compile commands:

nvcc -arch=sm_75 -rdc=true -lcudadevrt -I . models/phylogenetics/crbd/crbd_webppl.cu -o smc.exe -std=c++11 -O3
g++ -x c++ -I . models/phylogenetics/crbd/crbd_webppl.cu -o smc.exe -std=c++11 -O3
*/

// Bisse-32 tree
// This model on local WebPPL with 10000 particles took ~42 sec
// This program on CPU took ~0.11 sec
// This program on GPU took ~0.265 sec

// Primate tree tree
// This model on local WebPPL with 10000 particles took ~323 sec
// This program on CPU took ~1.04 sec
// This program on GPU took ~0.606 sec

typedef short treeIdx_t;
struct progState_t {
    treeIdx_t treeIdx;
};
typedef primate_tree_t tree_t;

BBLOCK_HELPER_DECLARE(crbdGoesUndetected, progState_t, bool);

const int MAX_DIV = 5;
const int MAX_LAM = 5;

#define NUM_BBLOCKS 2
INIT_GLOBAL(progState_t, NUM_BBLOCKS)

BBLOCK_DATA(tree, tree_t, 1)

BBLOCK_DATA(lambda, floating_t, 1) // prolly faster to just pass these as args... they should be generated in particle anyway?
BBLOCK_DATA(mu, floating_t, 1)
BBLOCK_DATA(rho, floating_t, 1)


void initCBD() {
    // lambda ~ gamma( 1.0, 1.0 )
    // mu     ~ gamma( 1.0, 1.0 )
    *lambda = 0.2; // birth rate
    *mu = 0.1; // death rate
    *rho = 1.0;

    COPY_DATA_GPU(tree, tree_t, 1)
    COPY_DATA_GPU(lambda, floating_t, 1)
    COPY_DATA_GPU(mu, floating_t, 1)
    COPY_DATA_GPU(rho, floating_t, 1)

}

BBLOCK_HELPER(M_crbdGoesUndetected, {

    if(max_M == 0) {
        printf("Aborting crbdGoesUndetected simulation, too deep!\n");
        return 0; // What do return instead of NaN?
    }

    if(! BBLOCK_CALL(crbdGoesUndetected<T>, startTime) && ! BBLOCK_CALL(crbdGoesUndetected<T>, startTime))
        return 1;
    else
        return 1 + BBLOCK_CALL(M_crbdGoesUndetected, startTime, max_M - 1);

}, int, floating_t startTime, int max_M)

BBLOCK_HELPER(crbdGoesUndetected, {

    floating_t lambdaLocal = *DATA_POINTER(lambda);
    floating_t muLocal = *DATA_POINTER(mu);
    floating_t rhoLocal = *DATA_POINTER(rho);

    // extreme values patch 1/2
    if (lambdaLocal - muLocal > MAX_DIV)
        return false;
    
    if (lambdaLocal == 0.0) {
        return ! SAMPLE(bernoulli, rhoLocal);
        /*
        if (flip(rhoLocal))
            return false
        else
            return true
        */
    }
    // end extreme values patch 1/2

    floating_t t = SAMPLE(exponential, lambdaLocal + muLocal);
    
    floating_t currentTime = startTime - t;
    if(currentTime < 0)
        return ! SAMPLE(bernoulli, rhoLocal);
    
    bool speciation = SAMPLE(bernoulli, lambdaLocal / (lambdaLocal + muLocal));
    if (! speciation)
        return true;
    
    return BBLOCK_CALL(crbdGoesUndetected<T>, currentTime) && BBLOCK_CALL(crbdGoesUndetected<T>, currentTime);

}, bool, floating_t startTime)


BBLOCK_HELPER(simBranch, {

    floating_t lambdaLocal = *DATA_POINTER(lambda);

    // extreme values patch 2/2
	if (lambdaLocal > MAX_LAM) {
	    //console.log( "lambda: ", lambda )
	    return -INFINITY;
	}
	
	if (lambdaLocal == 0.0) {
	    // var t1 = startTime - stopTime
        return 0.0;
	}
	// extreme values patch 2/2

    floating_t t = SAMPLE(exponential, lambdaLocal);

    floating_t currentTime = startTime - t;

    if(currentTime <= stopTime)
        return 0.0;
    
    bool sideDetection = BBLOCK_CALL(crbdGoesUndetected<T>, currentTime);
    if(! sideDetection)
        return -INFINITY;
    
    return BBLOCK_CALL(simBranch<T>, currentTime, stopTime) + log(2.0);

}, floating_t, floating_t startTime, floating_t stopTime)


BBLOCK(simTree, progState_t, {

    tree_t* treeP = DATA_POINTER(tree);
    int treeIdx = PSTATE.treeIdx;

    int indexParent = treeP->idxParent[treeIdx];
    
    floating_t parentAge = treeP->ages[indexParent];
    floating_t age = treeP->ages[treeIdx];

    floating_t lnProb1 = - (*DATA_POINTER(mu)) * (parentAge - age);

    // Interior if at least one child
    bool interiorNode = treeP->idxLeft[treeIdx] != -1 || treeP->idxRight[treeIdx] != -1;
    floating_t lnProb2 = interiorNode ? log(*DATA_POINTER(lambda)) : log(*DATA_POINTER(rho));

    floating_t lnProb3 = BBLOCK_CALL(simBranch<progState_t>, parentAge, age);

    WEIGHT(lnProb1 + lnProb2 + lnProb3);

    // Instead of recurring, use pre-processed traversal order
    int nextIdx = treeP->idxNext[treeIdx];
    PSTATE.treeIdx = nextIdx;

    if(nextIdx == -1) {
        PC++;
        return;
    }

})

/*
BBLOCK(simCRBD1, progState_t, {
    tree_t* treeP = DATA_POINTER(tree);

    PSTATE.treeIdx = treeP->idxLeft[ROOT_IDX];

    int numLeaves = countLeaves(treeP->idxLeft, treeP->idxRight, NUM_NODES);
    floating_t corrFactor = (numLeaves - 1) * log(2.0) - lnFactorial(numLeaves);
    WEIGHT(corrFactor);

    PC++;
})

BBLOCK(simCRBD2, progState_t, {
    tree_t* treeP = DATA_POINTER(tree);

    const int MAX_M = 10000;
    int M = BBLOCK_CALL(M_crbdGoesUndetected, treeP->ages[PSTATE.treeIdx], MAX_M);
    WEIGHT(log(static_cast<floating_t>(M)));

    PC++;
})
*/

BBLOCK(simCRBD, progState_t, {

    tree_t* treeP = DATA_POINTER(tree);

    PSTATE.treeIdx = treeP->idxLeft[ROOT_IDX];

    int numLeaves = countLeaves(treeP->idxLeft, treeP->idxRight, treeP->NUM_NODES);
    floating_t corrFactor = (numLeaves - 1) * log(2.0) - lnFactorial(numLeaves);
    WEIGHT(corrFactor);
    // printf("corrFactor: %f\n", corrFactor);

    // Resample here perhaps?

    const int MAX_M = 10000;
    int M = BBLOCK_CALL(M_crbdGoesUndetected, treeP->ages[PSTATE.treeIdx], MAX_M);
    WEIGHT(log(static_cast<floating_t>(M)));
    // printf("log(M): %f\n", log(static_cast<floating_t>(M)));

    PC++;
    // BBLOCK_CALL(simTree);
})


MAIN(
    initCBD();
    
    INIT_BBLOCK(simCRBD, progState_t)
    // INIT_BBLOCK(simCRBD1, progState_t)
    // INIT_BBLOCK(simCRBD2, progState_t)
    INIT_BBLOCK(simTree, progState_t)

    SMC(progState_t, NULL)
)

