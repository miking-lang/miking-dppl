#include "hip/hip_runtime.h"
/*
 * File crbd_webppl.cu defines the constant rate birth death model 
 * as defined in WebPPL in the script linked to below. 
 * 
 * https://github.com/phyppl/probabilistic-programming/blob/master/webppl/phywppl/models/crbd.wppl
 *
 * This model traverses the tree with a pre-computed DFS path (defined by the next 
 * pointer in the tree) that corresponds to the recursive calls in the original model. 
 */

#include <stdio.h>

#include "inference/smc/smc.cuh"
#include "../tree-utils/tree_utils.cuh"
#include "utils/math.cuh"

/**
    This file traverses the tree with a precomputed DFS path that corresponds to the recursive calls. 
*/

typedef short treeIdx_t;
struct progState_t {
    floating_t lambda;
    floating_t mu;
    treeIdx_t treeIdx;
};
// typedef bisse32_tree_t tree_t;
// typedef bisse32precision_tree_t tree_t;
typedef primate_tree_t tree_t;

const int MAX_DIV = 5;
const int MAX_LAM = 5;

#define NUM_BBLOCKS 3
INIT_MODEL(progState_t, NUM_BBLOCKS)

BBLOCK_HELPER_DECLARE(crbdGoesUndetected, bool, floating_t, floating_t, floating_t);

BBLOCK_DATA(tree, tree_t, 1)

BBLOCK_DATA(rho, floating_t, 1)


void initCBD() {
    *rho = 1.0;

    COPY_DATA_GPU(rho, floating_t, 1)
}

BBLOCK_HELPER(M_crbdGoesUndetected, {

    if(maxM == 0) {
        printf("Aborting crbdGoesUndetected simulation, too deep!\n");
        return 1; // What do return instead of NaN?
    }

    // if(max_M < 9400)
        // printf("Max_M: %d\n", max_M);

    if(! BBLOCK_CALL(crbdGoesUndetected, startTime, lambda, mu) && ! BBLOCK_CALL(crbdGoesUndetected, startTime, lambda, mu))
        return 1;
    else
        return 1 + BBLOCK_CALL(M_crbdGoesUndetected, startTime, maxM - 1, lambda, mu);

}, int, floating_t startTime, int maxM, floating_t lambda, floating_t mu)

BBLOCK_HELPER(crbdGoesUndetected, {

    floating_t rhoLocal = *DATA_POINTER(rho);

    // extreme values patch 1/2
    if (lambda - mu > MAX_DIV)
        return false;
    
    if (lambda == 0.0) 
        return ! SAMPLE(bernoulli, rhoLocal);

    // end extreme values patch 1/2

    floating_t t = SAMPLE(exponential, lambda + mu);
    
    floating_t currentTime = startTime - t;
    if(currentTime < 0)
        return ! SAMPLE(bernoulli, rhoLocal);
    
    bool speciation = SAMPLE(bernoulli, lambda / (lambda + mu));
    if (! speciation)
        return true;
    
    return BBLOCK_CALL(crbdGoesUndetected, currentTime, lambda, mu) && BBLOCK_CALL(crbdGoesUndetected, currentTime, lambda, mu);

}, bool, floating_t startTime, floating_t lambda, floating_t mu)


BBLOCK_HELPER(simBranch, {

    // extreme values patch 2/2
	if (lambda > MAX_LAM) {
	    return -INFINITY;
	}
	
	if (lambda == 0.0) {
	    // var t1 = startTime - stopTime
        return 0.0;
	}
	// extreme values patch 2/2

    floating_t t = SAMPLE(exponential, lambda);

    floating_t currentTime = startTime - t;

    if(currentTime <= stopTime)
        return 0.0;
    
    bool sideDetection = BBLOCK_CALL(crbdGoesUndetected, currentTime, lambda, mu);
    if(! sideDetection)
        return -INFINITY;
    
    return BBLOCK_CALL(simBranch, currentTime, stopTime, lambda, mu) + log(2.0);

}, floating_t, floating_t startTime, floating_t stopTime, floating_t lambda, floating_t mu)


BBLOCK(simTree, {

    tree_t* treeP = DATA_POINTER(tree);
    int treeIdx = PSTATE.treeIdx;

    floating_t lambdaLocal = PSTATE.lambda;
    floating_t muLocal = PSTATE.mu;

    int indexParent = treeP->idxParent[treeIdx];
    
    floating_t parentAge = treeP->ages[indexParent];
    floating_t age = treeP->ages[treeIdx];

    floating_t lnProb1 = - muLocal * (parentAge - age);

    // Interior if at least one child
    bool interiorNode = treeP->idxLeft[treeIdx] != -1 || treeP->idxRight[treeIdx] != -1;
    floating_t lnProb2 = interiorNode ? log(lambdaLocal) : log(*DATA_POINTER(rho));

    floating_t lnProb3 = BBLOCK_CALL(simBranch, parentAge, age, lambdaLocal, muLocal);

    WEIGHT(lnProb1 + lnProb2 + lnProb3);

    // Instead of recurring, use pre-processed traversal order
    int nextIdx = treeP->idxNext[treeIdx];
    PSTATE.treeIdx = nextIdx;

    if(nextIdx == -1) {
        PC++;
        return;
    }

})


BBLOCK(simCRBD, {

    PSTATE.lambda = SAMPLE(gamma, 1.0, 1.0);
    floating_t epsilon = SAMPLE(uniform, 0.0, 1.0);
    PSTATE.mu = epsilon * PSTATE.lambda;

    tree_t* treeP = DATA_POINTER(tree);

    PSTATE.treeIdx = treeP->idxLeft[ROOT_IDX];

    int numLeaves = countLeaves(treeP->idxLeft, treeP->idxRight, treeP->NUM_NODES);
    floating_t corrFactor = (numLeaves - 1) * log(2.0) - lnFactorial(numLeaves);
    WEIGHT(corrFactor);

    PC++;
})

BBLOCK(survivorshipBias, {
    // Survivorship Bias, is done after simCRBD
    floating_t age = DATA_POINTER(tree)->ages[ROOT_IDX];
    int MAX_M = 10000;
    int M = BBLOCK_CALL(M_crbdGoesUndetected, age, MAX_M, PSTATE.lambda, PSTATE.mu);
    WEIGHT(log(static_cast<floating_t>(M)));
    PC++;
})


MAIN(
    initCBD();
    
    ADD_BBLOCK(simCRBD)
    ADD_BBLOCK(simTree)
    ADD_BBLOCK(survivorshipBias)

    SMC(NULL)
)

