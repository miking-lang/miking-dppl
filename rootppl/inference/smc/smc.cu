#include "hip/hip_runtime.h"
#ifndef SMC_IMPL_INCLUDED
#define SMC_IMPL_INCLUDED

/*
 * File smc_impl.cuh contains the implementation of the top-level SMC.
 */

#include <iostream>

#include "macros/macros.cuh"
#include "smc.cuh"
#include "dists/dists.cuh"
#include "particles_memory_handler.cuh"
#include "resample/systematic/sequential.cuh"
// #include "smc_include.cuh"

#ifdef __NVCC__
#include <hiprand/hiprand_kernel.h>
// #include "hip/hip_runtime_api.h"
#include "utils/cuda_error_utils.cuh"
#include "resample/systematic/parallel.cuh"
#include "smc_kernels.cuh"
#endif


double runSMC(const pplFunc_t* bblocks, int numBblocks, const int numParticles, const int particlesPerThread, 
                size_t progStateSize, callbackFunc_t callback, void* arg) {

    floating_t logNormConstant = 0;

    particles_t particles = allocateParticles(numParticles, progStateSize, false);
    
    #ifdef __NVCC__
    // Rather add an extra thread than add iterations for a few threads
    const int numThreads = (numParticles + particlesPerThread - 1) / particlesPerThread;

    const int NUM_BLOCKS_EXEC = (numThreads + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
    const int NUM_BLOCKS = (numParticles + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;

    hiprandState* randStates;
    cudaSafeCall(hipMalloc(&randStates, sizeof(hiprandState) * numThreads));
    initCurandStates<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(randStates, numThreads, 0);
    hipDeviceSynchronize();
    cudaCheckError();
    #endif

    resampler_t resampler = initResampler(numParticles, progStateSize);

    // hipProfilerStart();
    // Run program/inference
    while(true) {

        #ifdef __NVCC__
        execFuncs<<<NUM_BLOCKS_EXEC, NUM_THREADS_PER_BLOCK>>>(randStates, particles, bblocks, numParticles, numThreads, arg);
        hipDeviceSynchronize();
        cudaCheckError();
        floating_t logWeightSum = calcLogWeightSumPar(particles.weights, resampler, numParticles, NUM_BLOCKS, NUM_THREADS_PER_BLOCK);
        #else

        for(int i = 0; i < numParticles; i++) {
            int pc = particles.pcs[i];
            if(pc < numBblocks && pc >= 0)
                bblocks[pc](particles, i, arg);
        }
        floating_t logWeightSum = calcLogWeightSumSeq(particles.weights, resampler, numParticles);
        #endif

        logNormConstant += logWeightSum - log(numParticles);
        
        #ifdef __NVCC__
        resampleSystematicPar(particles, resampler, numParticles, NUM_BLOCKS);
        #else
        resampleSystematicSeq(particles, resampler, numParticles);
        #endif
        
        // This last resample increases variance perhaps? But convenient to not have to consider weights when extracting distribution. 
        if(particles.pcs[0] >= numBblocks) // Assumption: All terminate at the same time
            break;
        
    }
    // hipProfilerStop();

    if(callback != NULL)
        callback(particles, numParticles, NULL);

    // Clean up
    destResampler(resampler);
    freeParticles(particles);
    #ifdef __NVCC__
    cudaSafeCall(hipFree(randStates));
    #endif

    return logNormConstant;
}


void configureMemSizeGPU() {
    #ifdef __NVCC__

    // Read memory properties and define limits
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    size_t MAX_THREADS_RESIDENT = devProp.maxThreadsPerMultiProcessor * devProp.multiProcessorCount;
    size_t GPU_MEM_TOT = devProp.totalGlobalMem * 0.95; // Leave 5% of memory for global structures or just to be sure
    size_t GPU_MEM_HEAP = GPU_MEM_TOT * 0.20; // Arbitrarily set 20% of GPU memory to device allocated heap memory
    size_t GPU_MEM_STACK = GPU_MEM_TOT - GPU_MEM_HEAP;
    size_t MAX_LOCAL_MEM_PER_THREAD = 512000; // 512 KB on all compute capabilities according to CUDA docs
    size_t MAX_STACK_SIZE = min(MAX_LOCAL_MEM_PER_THREAD, GPU_MEM_STACK / MAX_THREADS_RESIDENT);
    MAX_STACK_SIZE *= 1.0; // For some reason, with nested inference, this limit must be lower. Also, lower can give better performance.
    
    // Set limits and read the resulting set limits
    size_t heapSize, stackSize;
    hipDeviceSetLimit(hipLimitMallocHeapSize, GPU_MEM_HEAP);
    hipDeviceSetLimit(hipLimitStackSize, MAX_STACK_SIZE);
    hipDeviceGetLimit(&heapSize, hipLimitMallocHeapSize);
    hipDeviceGetLimit(&stackSize, hipLimitStackSize);

    if(false) {
        std::cout << "Global Memory size: " << GPU_MEM_TOT / 1000000.0 << " MB" << std::endl;
        std::cout << "Stack per thread max size attempted to set: " << MAX_STACK_SIZE / 1000.0 << " KB" << std::endl;
        std::cout << "Stack per thread max size set: " << stackSize / 1000.0 << " KB" << std::endl;
        std::cout << "Device allocation heap max size: " << heapSize / 1000000.0 << " MB" << std::endl;
    }
    // cudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    #endif
}


#endif
