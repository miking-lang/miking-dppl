#include "hip/hip_runtime.h"
#ifndef SMC_IMPL_INCLUDED
#define SMC_IMPL_INCLUDED

/*
 * File smc_impl.cuh contains the implementation of the top-level SMC.
 */

#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <string>
#include <list> 
#include <tuple>

#ifdef _OPENMP
#include <omp.h>
#endif

#include "file_handler.cuh"
#include "macros/macros.cuh"
#include "smc.cuh"
#include "dists/dists.cuh"
#include "particles_memory_handler.cuh"
#include "resample/systematic/systematic_cpu.cuh"

#ifdef __NVCC__
#include <hiprand/hiprand_kernel.h>
// #include "hip/hip_runtime_api.h"
#include "utils/cuda_error_utils.cuh"
#include "resample/systematic/systematic_gpu.cuh"
#include "smc_kernels.cuh"
#endif

// Resample if relative ESS < ESS_THRESHOLD * N (default threshold in Birch is 0.7) 
#ifndef ESS_THRESHOLD
const floating_t ESS_THRESHOLD = 0.7;
#endif
 
double runSMC(const pplFunc_t firstBblock, const int numParticles, const int ompThreads, const int particlesPerThread,
                size_t progStateSize, callbackFunc_t callback, void* arg) {

    #ifdef _OPENMP
    if(ompThreads > 0)
        omp_set_num_threads(ompThreads);
    #endif
    floating_t logNormConstant = 0;

    particles_t particles = allocateParticles(numParticles, progStateSize, false);
    
    #ifdef __NVCC__
    // Rather add an extra thread than add iterations for a few threads
    const int numThreads = (numParticles + particlesPerThread - 1) / particlesPerThread;

    const int NUM_BLOCKS_EXEC = (numThreads + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;
    const int NUM_BLOCKS = (numParticles + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;

    hiprandState* randStates;
    cudaSafeCall(hipMalloc(&randStates, sizeof(hiprandState) * numThreads));
    initCurandStates<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(randStates, numThreads, 0);
    hipDeviceSynchronize();
    cudaCheckError();
    #endif

    resampler_t resampler = initResampler(numParticles, progStateSize);
    std::list<double> essList;

    // Set all next pointers to first bblock
    #ifdef __NVCC__
    initParticlesNext<<<NUM_BLOCKS_EXEC, NUM_THREADS_PER_BLOCK>>>(particles, numParticles, firstBblock);
    #else
    #pragma omp parallel for
    for(int i = 0; i < numParticles; i++) {
        particles.next[i] = firstBblock;
    }
    #endif

    // Run program/inference
    while(true) {

        #ifdef __NVCC__
        execFuncs<<<NUM_BLOCKS_EXEC, NUM_THREADS_PER_BLOCK>>>(randStates, particles, numParticles, numThreads, arg);
        hipDeviceSynchronize();
        cudaCheckError();
        floating_t logWeightSum, ess;
        std::tie(logWeightSum, ess) = calcLogWeightSumAndESSGpu(particles.weights, resampler, numParticles, NUM_BLOCKS, NUM_THREADS_PER_BLOCK);
        
        hipDeviceSynchronize();
        #else

        #pragma omp parallel for
        for(int i = 0; i < numParticles; i++) {
            pplFunc_t next = particles.next[i];
            if(next != NULL)
                next(particles, i, arg);
        }
        floating_t logWeightSum, ess;
        std::tie(logWeightSum, ess) = calcLogWeightSumAndESSCpu(particles.weights, resampler, numParticles);

        #endif

        essList.push_back(ess);

        if(logWeightSum == -INFINITY) {
            printf("logWeightSum is -INFINITY, terminating...\n");
            break;
        }
        
        if (isnan(logWeightSum)) {
            printf("Weight Sum is NaN, terminating...\n");
            break;
        }

        // Assumption: All terminate at the same time
        bool terminate = particles.next[0] == NULL;

        // Break above assumption: Only terminate if all particles have terminated
        if (terminate) {
            for(int i = 0; i < numParticles; i++) {
                bool particleTerminated = particles.next[i] == NULL;
                if (! particleTerminated) {
                    terminate = false;
                    break;
                }
            }
        }

        bool resample = ess < ESS_THRESHOLD * numParticles;

        // Only add to log norm constant if resampling should be done (or if we are about to terminate)
        if (resample || terminate)
            logNormConstant += logWeightSum - log(numParticles);

        // Resampling will be skipped the last SMC iteration. Instead, weights will be renormalised and logged so they represent log-probabilities.
        if(terminate) {
            #ifdef __NVCC__
            normaliseWeightsGpu(particles.weights, logWeightSum, numParticles, NUM_BLOCKS, NUM_THREADS_PER_BLOCK);
            #else
            normaliseWeightsCpu(particles.weights, logWeightSum, numParticles);
            #endif

            break;
        }
        
        if (resample) {
            #ifdef __NVCC__
            resampleSystematicGpu(particles, resampler, numParticles, NUM_BLOCKS);
            #else
            resampleSystematicCpu(particles, resampler, numParticles);
            #endif
        }
        
    }

    printf("%f\n", logNormConstant);
    writeLogNormConstToFile(logNormConstant);
    writeESSToFile(essList);

    if(callback != NULL)
        callback(particles, numParticles, NULL);

    // Clean up
    destResampler(resampler);
    freeParticles(particles);
    #ifdef __NVCC__
    cudaSafeCall(hipFree(randStates));
    #endif

    return logNormConstant;
}

void prepareSMC() {
    configureMemSizeGPU();
    prepareFile(Z_FILE_NAME, true);
    prepareFile(ESS_FILE_NAME, true);
}

void finishFilesSMC() {
    finishFile(Z_FILE_NAME, true);
    finishFile(ESS_FILE_NAME, false);
}


void configureMemSizeGPU() {
    #ifdef __NVCC__

    // Read memory properties and define limits
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    size_t MAX_THREADS_RESIDENT = devProp.maxThreadsPerMultiProcessor * devProp.multiProcessorCount;
    size_t GPU_MEM_TOT = devProp.totalGlobalMem * 0.95; // Leave 5% of memory for global structures or just to be sure
    size_t GPU_MEM_HEAP = GPU_MEM_TOT * 0.20; // Arbitrarily set 20% of GPU memory to device allocated heap memory
    size_t GPU_MEM_STACK = GPU_MEM_TOT - GPU_MEM_HEAP;
    size_t MAX_LOCAL_MEM_PER_THREAD = 512000; // 512 KB on all compute capabilities according to CUDA docs
    size_t MAX_STACK_SIZE = min(MAX_LOCAL_MEM_PER_THREAD, GPU_MEM_STACK / MAX_THREADS_RESIDENT);
    MAX_STACK_SIZE *= 1.0; // For some reason, with nested inference, this limit must be lower. Also, lower can give better performance.
    
    // Set limits and read the resulting set limits
    size_t heapSize, stackSize;
    hipDeviceSetLimit(hipLimitMallocHeapSize, GPU_MEM_HEAP);
    hipDeviceSetLimit(hipLimitStackSize, MAX_STACK_SIZE);
    hipDeviceGetLimit(&heapSize, hipLimitMallocHeapSize);
    hipDeviceGetLimit(&stackSize, hipLimitStackSize);

    if(false) {
        std::cout << "Global Memory size: " << GPU_MEM_TOT / 1000000.0 << " MB" << std::endl;
        std::cout << "Stack per thread max size attempted to set: " << MAX_STACK_SIZE / 1000.0 << " KB" << std::endl;
        std::cout << "Stack per thread max size set: " << stackSize / 1000.0 << " KB" << std::endl;
        std::cout << "Device allocation heap max size: " << heapSize / 1000000.0 << " MB" << std::endl;
    }
    // cudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    #endif
}


#endif
