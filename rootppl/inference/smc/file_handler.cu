#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <sstream>
#include <list> 
#include <iterator> 

#include "file_handler.cuh"


void prepareFile(std::string fileName, bool truncate) {
    std::ofstream resFile (fileName, (truncate ? std::ofstream::out | std::ofstream::trunc : std::ios_base::app));
    resFile << "[";
    resFile.close();
}

void finishFile(std::string fileName, bool removeLast) {
    std::ifstream resFile1 (fileName);
    std::stringstream buffer;
    buffer << resFile1.rdbuf();
    std::string contents = buffer.str();
    resFile1.close();
    if(removeLast)
        contents.pop_back();
    contents += "]\n";
    std::ofstream resFile2 (fileName,  std::ofstream::out | std::ofstream::trunc);
    resFile2 << contents;
    resFile2.close();
}

void writeLogNormConstToFile(double logNormConstant) {
    // std::ofstream resFile (fileName);
    std::ofstream resFile (Z_FILE_NAME, std::ios_base::app); // If append to file is wanted
    if(resFile.is_open()) {

        resFile << logNormConstant << ",";
        resFile.close();
    } else {
        printf("Could not open file %s\n", Z_FILE_NAME.c_str());
    }
}


void writeESSToFile(std::list<double> essList) {
    // std::ofstream resFile (fileName);
    std::ofstream resFile (ESS_FILE_NAME, std::ios_base::app); // If append to file is wanted
    if(resFile.is_open()) {
        resFile << "[";
        std::list <double> :: iterator it;
        for(it = essList.begin(); it != essList.end(); ++it) 
            resFile << *it << " ";
        resFile << "],\n";
        resFile.close();
    } else {
        printf("Could not open file %s\n", ESS_FILE_NAME.c_str());
    }
}
