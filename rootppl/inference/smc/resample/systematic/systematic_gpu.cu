
#include <hip/hip_runtime.h>

/*
 * File systematic_gpu.cuh contains the GPU implementation of the systematic resampling. 
 * This implementation is inspired by the paper by L. M. Murray et. al.: 
 * Parallel resampling in the particle filter https://arxiv.org/abs/1301.4019
 */

#ifdef __NVCC__

#include "inference/smc/resample/common.cuh"
#include "kernels.cuh"
#include "utils/cuda_error_utils.cuh"
#include "systematic_gpu.cuh"

#include <curand_kernel.h>
#include <thrust/scan.h>

HOST DEV void prefixSumNaive(floating_t* w, resampler_t resampler, int numParticles) {
    resampler.prefixSum[0] = w[0];
    for(int i = 1; i < numParticles; i++)
        resampler.prefixSum[i] = resampler.prefixSum[i-1] + w[i];
}

HOST std::tuple<floating_t, floating_t> calcLogWeightSumAndESSGpu(floating_t* w, resampler_t& resampler, int numParticles, int numBlocks, int numThreadsPerBlock) {

    floating_t maxLogWeight = *(thrust::max_element(thrust::device, w, w + numParticles));
    resampler.maxLogWeight = maxLogWeight;
    // floating_t maxLogWeight = maxNaive(w, numParticles);
    
    scaleExpWeightsAndSquareWeightsKernel<<<numBlocks, numThreadsPerBlock>>>(w, numParticles, maxLogWeight, resampler.wSquared);
    cudaDeviceSynchronize();
    thrust::inclusive_scan(thrust::device, w, w + numParticles, resampler.prefixSum); // prefix sum
    // prefixSumNaive(w, resampler, numParticles);

    // At this point: w are scaled weights (not log), prefixSum[numParticles-1] is the scaled sum
    floating_t ess = calcESSHelperGpu(w, resampler.prefixSum[numParticles - 1], resampler.wSquared, numParticles);

    renormaliseKernel<<<numBlocks, numThreadsPerBlock>>>(w, resampler.prefixSum, numParticles, maxLogWeight);
    
    cudaDeviceSynchronize();
    // return resampler.prefixSum[numParticles - 1];
    return std::make_tuple(resampler.prefixSum[numParticles - 1], ess);
}

HOST floating_t calcESSHelperGpu(floating_t* scaledW, floating_t scaledWeightSum, floating_t* scaledWSquared, int numParticles) {

    // Kernel saving new square exp log weights
    // expSquareWeightsKernel<<<numBlocks, numThreadsPerBlock>>>(w, resampler.wSquared, resampler.maxLogWeight, numParticles);

    // Thrust for summing squared weights
    cudaDeviceSynchronize();
    floating_t wSumOfSquares = (thrust::reduce(thrust::device, scaledWSquared, scaledWSquared + numParticles));

    floating_t wSumSquared = scaledWeightSum * scaledWeightSum;

    return wSumSquared / wSumOfSquares;
}


HOST DEV void decideAncestors(resampler_t& resampler, floating_t u, int numParticles, int numBlocks, int numThreadsPerBlock) {

    systematicCumulativeOffspringKernel<<<numBlocks, numThreadsPerBlock>>>(resampler.prefixSum, resampler.cumulativeOffspring, u, numParticles);

    cumulativeOffspringToAncestorKernel<<<numBlocks, numThreadsPerBlock>>>(resampler.cumulativeOffspring, resampler.ancestor, numParticles);

}

HOST DEV void postUniform(particles_t& particles, resampler_t& resampler, floating_t u, int numParticles, int numBlocks, int numThreadsPerBlock) {

    decideAncestors(resampler, u, numParticles, numBlocks, numThreadsPerBlock);

    // Copy states
    copyStatesKernel<<<numBlocks, numThreadsPerBlock>>>(resampler.auxParticles, particles, resampler.ancestor, numParticles, resampler.progStateSize);
    cudaDeviceSynchronize();

    // Swap pointers
    particles_t tempAux = resampler.auxParticles;
    resampler.auxParticles = particles;
    particles = tempAux;
}

DEV void resampleSystematicGpuNested(curandState* randState, particles_t& particles, resampler_t& resampler, int numParticles, int numBlocks) {
    
    floating_t u = uniform(randState, 0.0f, 1.0f);
    
    postUniform(particles, resampler, u, numParticles, numBlocks, NUM_THREADS_PER_BLOCK_NESTED);
}

void resampleSystematicGpu(particles_t& particles, resampler_t& resampler, int numParticles, int numBlocks) {

    floating_t u = uniformCPU(generatorRes);

    postUniform(particles, resampler, u, numParticles, numBlocks, NUM_THREADS_PER_BLOCK);
}

void normaliseWeightsGpu(floating_t* w, floating_t logWeightSum, int numParticles, int numBlocks, int numThreadsPerBlock) {
    normaliseWeightsKernel<<<numBlocks, numThreadsPerBlock>>>(w, logWeightSum, numParticles);
    cudaDeviceSynchronize();
}


#endif