
#include <hip/hip_runtime.h>

/*
 * File smc_kernels.cu contains definitions of kernels used by SMC. 
 */

 #ifdef __NVCC__

 #include <curand_kernel.h>
 #include "inference/smc/smc.cuh"
 #include "smc_kernels.cuh"

__global__ void initCurandStates(curandState* randStates, int numThreads, int seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= numThreads || i < 0) return;

    // Double check this seed, need only to be unique over one inference, as time should vary between inferences. 
    // curand_init(1234 + clock64(), seed * numThreads + i, 0, &particles->randStates[i]);
    curandState randStateLocal = randStates[i];
    curand_init(1234 + clock64(), seed * numThreads + i, 0, &randStateLocal);
    randStates[i] = randStateLocal;
}

__global__ void execFuncs(curandState* randStates, particles_t particles, const pplFunc_t* funcs, 
                            int numParticles, int numThreads, void* arg) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if(i >= numParticles || i < 0) return;
    if(i >= numThreads || i < 0) return;

    curandState randStateLocal = randStates[i];
    
    for(int j = i; j < numParticles; j += numThreads) {
        // printf("j: %d\n", j);
        // funcs[particles.pcs[i]](&randStateLocal, particles, i, arg);
        funcs[particles.pcs[j]](&randStateLocal, particles, j, arg);
    }


    randStates[i] = randStateLocal;
}

#endif
